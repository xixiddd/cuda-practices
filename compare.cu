#include <cstdio>
#include <hip/hip_runtime.h>

#define RADIUS                3
#define THREADS_PER_BLOCK     512

__global__ void windowSumNaiveKernel(const float* A, float* B, int n) {
  int out_index = blockDim.x * blockIdx.x + threadIdx.x;
  int in_index = out_index + RADIUS;
  if (out_index < n) {
    float sum = 0.;
#pragma unroll
    for (int i = -RADIUS; i <= RADIUS; ++i) {
      sum += A[in_index + i];
    }
    B[out_index] = sum;
  }
}

__global__ void windowSumKernel(const float* A, float* B, int n) {
  __shared__ float temp[THREADS_PER_BLOCK + 2 * RADIUS];
  int out_index = blockDim.x * blockIdx.x + threadIdx.x;
  int in_index = out_index + RADIUS;
  int local_index = threadIdx.x + RADIUS;
  if (out_index < n) {
    temp[local_index] = A[in_index];
    if (threadIdx.x < RADIUS) {
      temp[local_index - RADIUS] = A[in_index - RADIUS];
      temp[local_index + THREADS_PER_BLOCK] = A[in_index +  THREADS_PER_BLOCK];
    }
    __syncthreads();
    float sum = 0.;
#pragma unroll
    for (int i = -RADIUS; i <= RADIUS; ++i) {
      sum += temp[local_index + i];
    }
    B[out_index] = sum;
  }
}

void windowSumNaive(const float* A, float* B, int n) {
    float *d_A, *d_B;
    int size = n * sizeof(float);
    hipMalloc((void **) &d_A, (n + 2 * RADIUS) * sizeof(float));
    hipMemset(d_A, 0, (n + 2 * RADIUS) * sizeof(float));
    hipMemcpy(d_A + RADIUS, A, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, size);
    dim3 threads(THREADS_PER_BLOCK, 1, 1);
    dim3 blocks((n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1, 1);
    windowSumNaiveKernel<<<blocks, threads>>>(d_A, d_B, n);
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
}

void windowSum(const float* A, float* B, int n) {
    float *d_A, *d_B;
    int size = n * sizeof(float);
    hipMalloc((void **) &d_A, (n + 2 * RADIUS) * sizeof(float));
    hipMemset(d_A, 0, (n + 2 * RADIUS) * sizeof(float));
    hipMemcpy(d_A + RADIUS, A, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, size);
    dim3 threads(THREADS_PER_BLOCK, 1, 1);
    dim3 blocks((n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1, 1);
    windowSumKernel<<<blocks, threads>>>(d_A, d_B, n);
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
}

int main() {
  int n = 1024 * 1024;
  float* A = new float[n];
  float* B = new float[n];
  for (int i = 0; i < n; ++i) {
    A[i] = i;
  }
  
  hipEvent_t start, stop;
  float elapsedTime = 0.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  windowSumNaive(A, B, n);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("windowSumNaive: %f ms\n", elapsedTime);
  // cudaEventDestroy(start);
  // cudaEventDestroy(stop);
  
  // cudaEvent_t start, stop;
  elapsedTime = 0.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  windowSum(A, B, n);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("windowSum: %f ms\n", elapsedTime);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  delete [] A;
  delete [] B;
  return 0;
}
